#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>


#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/Parallel.h>

#define BLOCK 16

__forceinline__ __device__ bool within_bounds(int h, int w, int H, int W) {
  return h >= 0 && h < H && w >= 0 && w < W;
}

template <typename scalar_t>
__global__ void corr_index_forward_kernel(
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> volume,
    const torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> coords,
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> corr,
    int r)
{
  // batch index
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int n = blockIdx.z;

  const int h1 = volume.size(1);
  const int w1 = volume.size(2);
  const int h2 = volume.size(3);
  const int w2 = volume.size(4);

  if (!within_bounds(y, x, h1, w1)) {
    return;
  }

  float x0 = coords[n][0][y][x];
  float y0 = coords[n][1][y][x];

  float dx = x0 - floor(x0);
  float dy = y0 - floor(y0);

  int rd = 2*r + 1;
  for (int i=0; i<rd+1; i++) {
    for (int j=0; j<rd+1; j++) {
      int x1 = static_cast<int>(floor(x0)) - r + i;
      int y1 = static_cast<int>(floor(y0)) - r + j;

      if (within_bounds(y1, x1, h2, w2)) {
        scalar_t s = volume[n][y][x][y1][x1];

        if (i > 0 && j > 0)
          corr[n][i-1][j-1][y][x] += s * scalar_t(dx * dy);

        if (i > 0 && j < rd)
          corr[n][i-1][j][y][x] += s * scalar_t(dx * (1.0f-dy));

        if (i < rd && j > 0)
          corr[n][i][j-1][y][x] += s * scalar_t((1.0f-dx) * dy);

        if (i < rd && j < rd)
          corr[n][i][j][y][x] += s * scalar_t((1.0f-dx) * (1.0f-dy));

      }
    }
  }
}


template <typename scalar_t>
__global__ void corr_index_backward_kernel(
    const torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> coords,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> corr_grad,
    torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> volume_grad,
    int r)
{
  // batch index
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int n = blockIdx.z;

  const int h1 = volume_grad.size(1);
  const int w1 = volume_grad.size(2);
  const int h2 = volume_grad.size(3);
  const int w2 = volume_grad.size(4);

  if (!within_bounds(y, x, h1, w1)) {
    return;
  }

  float x0 = coords[n][0][y][x];
  float y0 = coords[n][1][y][x];

  float dx = x0 - floor(x0);
  float dy = y0 - floor(y0);

  int rd = 2*r + 1;
  for (int i=0; i<rd+1; i++) {
    for (int j=0; j<rd+1; j++) {
      int x1 = static_cast<int>(floor(x0)) - r + i;
      int y1 = static_cast<int>(floor(y0)) - r + j;

      if (within_bounds(y1, x1, h2, w2)) {
        scalar_t g = 0.0;
        if (i > 0 && j > 0)
          g += corr_grad[n][i-1][j-1][y][x] * scalar_t(dx * dy);

        if (i > 0 && j < rd)
          g += corr_grad[n][i-1][j][y][x] * scalar_t(dx * (1.0f-dy));

        if (i < rd && j > 0)
          g += corr_grad[n][i][j-1][y][x] * scalar_t((1.0f-dx) * dy);

        if (i < rd && j < rd)
          g += corr_grad[n][i][j][y][x] * scalar_t((1.0f-dx) * (1.0f-dy));

        volume_grad[n][y][x][y1][x1] += g;
      }
    }
  }
}

std::vector<torch::Tensor> corr_index_cuda_forward(
    torch::Tensor volume,
    torch::Tensor coords,
    int radius)
{
  const auto batch_size = volume.size(0);
  const auto ht = volume.size(1);
  const auto wd = volume.size(2);

  const dim3 blocks((wd + BLOCK - 1) / BLOCK, 
                    (ht + BLOCK - 1) / BLOCK, 
                    batch_size);
  
  const dim3 threads(BLOCK, BLOCK);

  auto opts = volume.options();
  torch::Tensor corr = torch::zeros(
    {batch_size, 2*radius+1, 2*radius+1, ht, wd}, opts);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(volume.type(), "sampler_forward_kernel", ([&] {
    corr_index_forward_kernel<scalar_t><<<blocks, threads>>>(
      volume.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
      coords.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
      corr.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
      radius);
   }));

  return {corr};

}

std::vector<torch::Tensor> corr_index_cuda_backward(
  torch::Tensor volume,
  torch::Tensor coords,
  torch::Tensor corr_grad,
  int radius)
{
  const auto batch_size = volume.size(0);
  const auto ht = volume.size(1);
  const auto wd = volume.size(2);

  auto volume_grad = torch::zeros_like(volume);

  const dim3 blocks((wd + BLOCK - 1) / BLOCK, 
                    (ht + BLOCK - 1) / BLOCK, 
                    batch_size);

  const dim3 threads(BLOCK, BLOCK);


  AT_DISPATCH_FLOATING_TYPES_AND_HALF(volume.type(), "sampler_backward_kernel", ([&] {
    corr_index_backward_kernel<scalar_t><<<blocks, threads>>>(
      coords.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
      corr_grad.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
      volume_grad.packed_accessor32<scalar_t,5,torch::RestrictPtrTraits>(),
      radius);
   }));

  return {volume_grad};
}